#include "hip/hip_runtime.h"
#include "optixParams.h" // our launch params


extern "C" {
    __constant__ LaunchParams optixLaunchParams;
}

// ray types
enum { RAIDANCE=0, SHADOW, RAY_TYPE_COUNT };

struct RadiancePRD{
    float3   emitted;
    float3   radiance;
    float3   attenuation;
    float3   origin;
    float3   direction;
    bool done;
    uint32_t seed;
    int32_t  countEmitted;
} ;

struct shadowPRD{
    float shadowAtt;
    uint32_t seed;
} ;




// -------------------------------------------------------

extern "C" __global__ void __closesthit__radiance() {

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    RadiancePRD &prd = *(RadiancePRD *)getPRD<RadiancePRD>();

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];

    const float3 nn = normalize(make_float3(n));
    // intersection position
    const float3 &rayDir =  optixGetWorldRayDirection();
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDir ;


    if (prd.countEmitted && length(sbtData.emission) != 0) {
        prd.emitted = sbtData.emission ;
        return;
    }
    else
        prd.emitted = make_float3(0.0f);

    uint32_t seed = prd.seed;

    {
        const float z1 = rnd(seed);
        const float z2 = rnd(seed);

        float3 w_in;
        cosine_sample_hemisphere( z1, z2, w_in );
        Onb onb( nn );
        onb.inverse_transform( w_in );
        prd.direction = w_in;
        prd.origin    = pos;

        prd.attenuation *= sbtData.diffuse ;
        prd.countEmitted = false;
    }
    

    const float z1 = rnd(seed);
    const float z2 = rnd(seed);
    prd.seed = seed;


    const float3 lightV1 = make_float3(0.47f, 0.0, 0.0f);
    const float3 lightV2 = make_float3(0.0f, 0.0, 0.38f);
    const float3 light_pos = make_float3(optixLaunchParams.global->lightPos) + lightV1 * z1 + lightV2 * z2;

    // Calculate properties of light sample (for area based pdf)
    const float  Ldist = length(light_pos - pos );
    const float3 L     = normalize(light_pos - pos );
    const float  nDl   = dot( nn, L );
    const float3 Ln    = normalize(cross(lightV1, lightV2));
    const float  LnDl  = -dot( Ln, L );

    float weight = 0.0f;
    if( nDl > 0.0f && LnDl > 0.0f )
    {
        uint32_t occluded = 0u;
        optixTrace(optixLaunchParams.traversable,
            pos,
            L,
            0.001f,         // tmin
            Ldist - 0.01f,  // tmax
            0.0f,                    // rayTime
            OptixVisibilityMask( 1 ),
            OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
            SHADOW,      // SBT offset
            RAY_TYPE_COUNT,          // SBT stride
            SHADOW,      // missSBTIndex
            occluded);

        if( !occluded )
        {
            const float att = Ldist * Ldist;
            const float A = length(cross(lightV1, lightV2));
            weight = nDl * LnDl * A  / att;
        }
    }

    prd.radiance += make_float3(5.0f, 5.0f, 5.0f) * weight * optixLaunchParams.global->lightScale;
}


extern "C" __global__ void __anyhit__radiance() {

}


// miss sets the background color
extern "C" __global__ void __miss__radiance() {

    RadiancePRD &prd = *(RadiancePRD*)getPRD<RadiancePRD>();
    // set black as background color
    prd.radiance = make_float3(0.0f, 0.0f, 0.0f);
    prd.done = true;
}


// -----------------------------------------------
// Shadow rays

extern "C" __global__ void __closesthit__shadow() {

    optixSetPayload_0( static_cast<uint32_t>(true));
}


// any hit for shadows
extern "C" __global__ void __anyhit__shadow() {

}


// miss for shadows
extern "C" __global__ void __miss__shadow() {

    optixSetPayload_0( static_cast<uint32_t>(false));
}



// -----------------------------------------------
// Primary Rays


extern "C" __global__ void __raygen__renderFrame() {

    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const auto &camera = optixLaunchParams.camera;  

    const int &maxDepth = optixLaunchParams.frame.maxDepth;
 
    float squaredRaysPerPixel = float(optixLaunchParams.frame.raysPerPixel);
    float2 delta = make_float2(1.0f/squaredRaysPerPixel, 1.0f/squaredRaysPerPixel);

    float3 result = make_float3(0.0f);

    uint32_t seed = tea<4>( ix * optixGetLaunchDimensions().x + iy, optixLaunchParams.frame.frame );

    for (int i = 0; i < squaredRaysPerPixel; ++i) {
        for (int j = 0; j < squaredRaysPerPixel; ++j) {

            const float2 subpixel_jitter = make_float2( delta.x * (i + rnd(seed)), delta.y * (j + rnd( seed )));
            const float2 screen(make_float2(ix + subpixel_jitter.x, iy + subpixel_jitter.y)
                            / make_float2(optixGetLaunchDimensions().x, optixGetLaunchDimensions().y) * 2.0 - 1.0);
        
            // note: nau already takes into account the field of view and ratio when computing 
            // camera horizontal and vertical
            float3 origin = camera.position;
            float3 rayDir = normalize(camera.direction
                                + (screen.x ) * camera.horizontal
                                + (screen.y ) * camera.vertical);

            RadiancePRD prd;
            prd.emitted      = make_float3(0.f);
            prd.radiance     = make_float3(0.f);
            prd.attenuation  = make_float3(1.f);
            prd.countEmitted = true;
            prd.done         = false;
            prd.seed         = seed;

            uint32_t u0, u1;
            packPointer( &prd, u0, u1 );             
            
            for (int k = 0; k < maxDepth && !prd.done; ++k) {

                optixTrace(optixLaunchParams.traversable,
                        origin,
                        rayDir,
                        0.001f,    // tmin
                        1e20f,  // tmax
                        0.0f, OptixVisibilityMask( 1 ),
                        OPTIX_RAY_FLAG_NONE, RAIDANCE, RAY_TYPE_COUNT, RAIDANCE, u0, u1 );

                result += prd.emitted;
                result += prd.radiance * prd.attenuation;

                origin = prd.origin;
                rayDir = prd.direction;
            }
        }
    }

    result = result / (squaredRaysPerPixel*squaredRaysPerPixel);
    float gamma = optixLaunchParams.global->gamma;
    // compute index
    const uint32_t fbIndex = ix + iy*optixGetLaunchDimensions().x;

    optixLaunchParams.global->accumBuffer[fbIndex] = 
        (optixLaunchParams.global->accumBuffer[fbIndex] * optixLaunchParams.frame.subFrame +
        make_float4(result.x, result.y, result.z, 1)) /(optixLaunchParams.frame.subFrame+1);

    
    float4 rgbaf = optixLaunchParams.global->accumBuffer[fbIndex];
    //convert float (0-1) to int (0-255)
    const int r = int(255.0f*min(1.0f, pow(rgbaf.x, 1/gamma)));
    const int g = int(255.0f*min(1.0f, pow(rgbaf.y, 1/gamma)));
    const int b = int(255.0f*min(1.0f, pow(rgbaf.z, 1/gamma))) ;

    // convert to 32-bit rgba value 
    const uint32_t rgba = 0xff000000 | (r<<0) | (g<<8) | (b<<16);
    // write to output buffer
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
}
  

